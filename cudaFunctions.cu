#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
using namespace std;

#include "constants.h"
#include "cudaFunctions.h"

/**
 * Calculate the alignment score of 2 sequences.
 * @param int* pointer to array of symbols weights.
 * @param char* pointer to the first sequence.
 * @param char* pointer to the scecond sequence.
 * @param int the size of the second sequence.
 * @param int offset index.
 * @param int mutant index.
 * @return alignment score.
 */
__device__ int alignmentScoreGPU(int* symbolsWeights, const char* firstSeq, const char* secondSeq, int firstSeqSize, int secondSeqSize, int offset, int mutant)
{
    int i, row, col, afterMutant = 0, score = 0;

    for (i = 0; i <= secondSeqSize; i++)
    {
        if (i != mutant)
        {
            row = firstSeq[i + offset] - LETTER_A;
            col = secondSeq[i - afterMutant] - LETTER_A;
            score += symbolsWeights[row * ENGLISH_LETTERS + col];
        }
        else
            afterMutant = 1; // Reduce 1 from the index after the mutant
    }

    return score;
}

/**
 * Calculate all scores of 2 sequences.
 * @param int* pointer to array of symbols weights.
 * @param char* pointer to the first sequence.
 * @param char* pointer to the scecond sequence.
 * @param int the size of the first sequence.
 * @param int the size of the second sequence.
 * @param Score* pointer to array of scores.
 */
__global__ void calcAlignmentsScores(int* symbolsWeights, const char* firstSeq, const char* secondSeq, int firstSeqSize, int secondSeqSize, Score* allScores)
{
    int mutant = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.y * blockIdx.y + threadIdx.y;
    int score, diffOffset = firstSeqSize - secondSeqSize;

    if (mutant < secondSeqSize && offset <= diffOffset)
    {
        score = offset == firstSeqSize - secondSeqSize && mutant + 1 != secondSeqSize ? INT_MIN : 
                alignmentScoreGPU(symbolsWeights, firstSeq, secondSeq, firstSeqSize, secondSeqSize, offset, mutant + 1);
        allScores[offset * secondSeqSize + mutant] = { score, offset, mutant + 1 };
    }
}

/**
 * Check if cuda status success.
 * @param hipError_t* pointer to array of sequences.
 * @param int* pointer to array of symbols weights.
 * @param char* pointer to the first sequence.
 * @param char* pointer to sequence.
 * @param Score* pointer to array of scores.
 * @param string error message.
 * @return EXIT_SUCCESS if everything worked properly, EXIT_FAILURE else.
 */
int checkStatus(hipError_t cudaStatus, int* symbolsWeights, char* firstSeq, char* sequence, Score* allScores, string err)
{
    if (cudaStatus != hipSuccess)
    {
        cout << err << endl;

        hipFree(symbolsWeights);
        hipFree(firstSeq);
        hipFree(sequence);
        hipFree(allScores);

        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

int allAlignmentsScores(int* symbolsWeights, const char* firstSeq, char** sequences, Score** scoresPerSequence, int* numAllScores, int numOfSequences)
{
    int *devSymbolsWeights = 0, i;
    char *devFirstSeq = 0, *devSequence = 0;
    Score *devAllScores = 0;
    size_t firstSeqSize = strlen(firstSeq), seqSize, diffOffset, numOfScores;
    hipError_t cudaStatus;
    dim3 threadsPerBlock(NUM_OF_THREADS, NUM_OF_THREADS), blocksPerGrid;

    cudaStatus = hipMalloc((void**)&devSymbolsWeights, ENGLISH_LETTERS * ENGLISH_LETTERS * sizeof(int));
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda malloc failed on devSymbolsWeights!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMemcpy(devSymbolsWeights, symbolsWeights, ENGLISH_LETTERS * ENGLISH_LETTERS * sizeof(int), hipMemcpyHostToDevice);
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda memcpy failed on devSymbolsWeights!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMalloc((void**)&devFirstSeq, firstSeqSize * sizeof(char));
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda malloc failed on devFirstSeq!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMemcpy(devFirstSeq, firstSeq, firstSeqSize * sizeof(char), hipMemcpyHostToDevice);
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda memcpy failed on devFirstSeq!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    for (i = 0; i < numOfSequences; i++)
    {
        seqSize = strlen(sequences[i]);
        diffOffset = firstSeqSize - seqSize + 1;
        numOfScores = diffOffset * seqSize;
        blocksPerGrid.x = (numOfScores + threadsPerBlock.x - 1) / threadsPerBlock.x;
        blocksPerGrid.y = (numOfScores + threadsPerBlock.y - 1) / threadsPerBlock.y;

        cudaStatus = hipMalloc((void**)&devSequence, seqSize * sizeof(char));
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda malloc failed on devSequence!") == EXIT_FAILURE)
            return EXIT_FAILURE;

        cudaStatus = hipMemcpy(devSequence, sequences[i], seqSize * sizeof(char), hipMemcpyHostToDevice);
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda memcpy failed on devSequence!") == EXIT_FAILURE)
            return EXIT_FAILURE;

        cudaStatus = hipMalloc((void**)&devAllScores, numOfScores * sizeof(Score));
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda malloc failed on devAllScores!") == EXIT_FAILURE)
            return EXIT_FAILURE;

        calcAlignmentsScores<<<blocksPerGrid, threadsPerBlock>>>(devSymbolsWeights, devFirstSeq, devSequence, firstSeqSize, seqSize, devAllScores);
        cudaStatus = hipDeviceSynchronize();
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda kernel failed on calcAlignmentsScores!") == EXIT_FAILURE)
            return EXIT_FAILURE;

        numAllScores[i] = numOfScores; // Write the number of scores for each sequence
        scoresPerSequence[i] = (Score*)malloc(numAllScores[i] * sizeof(Score));
        if (!scoresPerSequence[i])
            return EXIT_FAILURE;
        
        cudaStatus = hipMemcpy(scoresPerSequence[i], devAllScores, numAllScores[i] * sizeof(Score), hipMemcpyDeviceToHost);
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda memcpy failed on scoresPerSequence!") == EXIT_FAILURE)
            return EXIT_FAILURE;

        cudaStatus = hipFree(devSequence);
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda free failed on devSequence!") == EXIT_FAILURE)
        return EXIT_FAILURE;

        cudaStatus = hipFree(devAllScores);
        if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda free failed on devAllScores!") == EXIT_FAILURE)
        return EXIT_FAILURE;
    }

    cudaStatus = hipFree(devSymbolsWeights);
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda free failed on devSymbolsWeights!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipFree(devFirstSeq);
    if (checkStatus(cudaStatus, devSymbolsWeights, devFirstSeq, devSequence, devAllScores, "Cuda free failed on devFirstSeq!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    return EXIT_SUCCESS;
}